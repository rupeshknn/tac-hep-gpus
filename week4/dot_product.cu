
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)


// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {

	// FIXME
	// Use atomicAdd
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N) {
	atomicAdd(C, A[idx]*B[idx]);
	}
}

int main() {
	
	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
	}

	*h_C = 0;


	// Allocate device memory
	hipMalloc(&d_A, DSIZE*sizeof(int));
	hipMalloc(&d_B, DSIZE*sizeof(int));
	hipMalloc(&d_C, sizeof(int));
	// Check memory allocation for errors
	cudaCheckErrors();
	// Copy the matrices on GPU
	hipMemcpy(d_A, h_A, DSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, DSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeof(int), hipMemcpyHostToDevice);
	// Check memory copy for errors
	cudaCheckErrors();
	// Define block/grid dimentions and launch kernel
	dot_product<<<DSIZE/BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_B, d_C, DSIZE);
	// Copy results back to host
	hipMemcpy(h_C, d_C, sizeof(int), hipMemcpyDeviceToHost);
    // Check copy for errors
	cudaCheckErrors();
	// Verify result
	printf("A.B = %d\n", *h_C);
	// Free allocated memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);
	
	return 0;

}