#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// Stencil kernel with shared memory
__global__ void stencilKernelShared(const int* A, int* Ac, int DSIZE, int radius) {
    extern __shared__ int shared[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int idx = blockIdx.x * blockDim.x + tx;
    int idy = blockIdx.y * blockDim.y + ty;

    int localIdx = tx + radius;
    int localIdy = ty + radius;

    // Copy to shared memory (with halo)
    if (idx < DSIZE && idy < DSIZE) {
        shared[localIdy * (blockDim.x + 2 * radius) + localIdx] = A[idy * DSIZE + idx];
    }

    // Load halo regions
    if (tx < radius) {
        if (idx >= radius) {
            shared[localIdy * (blockDim.x + 2 * radius) + tx] = A[idy * DSIZE + (idx - radius)];
        }
        if (idx + blockDim.x < DSIZE) {
            shared[localIdy * (blockDim.x + 2 * radius) + (localIdx + blockDim.x)] =
                A[idy * DSIZE + (idx + blockDim.x)];
        }
    }
    if (ty < radius) {
        if (idy >= radius) {
            shared[ty * (blockDim.x + 2 * radius) + localIdx] = A[(idy - radius) * DSIZE + idx];
        }
        if (idy + blockDim.y < DSIZE) {
            shared[(localIdy + blockDim.y) * (blockDim.x + 2 * radius) + localIdx] =
                A[(idy + blockDim.y) * DSIZE + idx];
        }
    }

    __syncthreads();

    // Apply stencil
    if (idx >= radius && idx < DSIZE - radius && idy >= radius && idy < DSIZE - radius) {
        int temp = -shared[localIdy * (blockDim.x + 2 * radius) + localIdx];
        for (int r = -radius; r <= radius; ++r) {
            temp += shared[(localIdy + r) * (blockDim.x + 2 * radius) + localIdx];
            temp += shared[localIdy * (blockDim.x + 2 * radius) + (localIdx + r)];
        }
        Ac[idy * DSIZE + idx] = temp;
    }
}

// Matrix multiplication kernel
__global__ void matmulKernel(const int* Ac, const int* Bc, int* C, int DSIZE) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < DSIZE && col < DSIZE) {
        int sum = 0;
        for (int k = 0; k < DSIZE; ++k) {
            sum += Ac[row * DSIZE + k] * Bc[k * DSIZE + col];
        }
        C[row * DSIZE + col] = sum;
    }
}

// Host function
int* stencilMatmul(bool isRand, int radius, const int DSIZE) {
    // Unified memory allocation
    int *A, *B, *Ac, *Bc, *C;
    CUDA_CHECK(hipMallocManaged(&A, DSIZE * DSIZE * sizeof(int)));
    CUDA_CHECK(hipMallocManaged(&B, DSIZE * DSIZE * sizeof(int)));
    CUDA_CHECK(hipMallocManaged(&Ac, DSIZE * DSIZE * sizeof(int)));
    CUDA_CHECK(hipMallocManaged(&Bc, DSIZE * DSIZE * sizeof(int)));
    CUDA_CHECK(hipMallocManaged(&C, DSIZE * DSIZE * sizeof(int)));

    // Initialize matrices
    for (int i = 0; i < DSIZE; ++i) {
        for (int j = 0; j < DSIZE; ++j) {
            A[i * DSIZE + j] = isRand ? rand() % 10 : 1;
            B[i * DSIZE + j] = isRand ? rand() % 10 : 1;
            Ac[i * DSIZE + j] = A[i * DSIZE + j];
            Bc[i * DSIZE + j] = B[i * DSIZE + j];
            C[i * DSIZE + j] = 0;
        }
    }

    // Kernel configurations
    dim3 blockDim(16, 16);
    dim3 gridDim((DSIZE + blockDim.x - 1) / blockDim.x, (DSIZE + blockDim.y - 1) / blockDim.y);
    int sharedMemSize = (blockDim.x + 2 * radius) * (blockDim.y + 2 * radius) * sizeof(int);

    // Create CUDA streams
    hipStream_t stream1, stream2;
    CUDA_CHECK(hipStreamCreate(&stream1));
    CUDA_CHECK(hipStreamCreate(&stream2));

    // Launch stencil kernels on different streams
    stencilKernelShared<<<gridDim, blockDim, sharedMemSize, stream1>>>(A, Ac, DSIZE, radius);
    stencilKernelShared<<<gridDim, blockDim, sharedMemSize, stream2>>>(B, Bc, DSIZE, radius);

    // Synchronize stencil streams
    CUDA_CHECK(hipStreamSynchronize(stream1));
    CUDA_CHECK(hipStreamSynchronize(stream2));

    // Launch matrix multiplication kernel
    matmulKernel<<<gridDim, blockDim>>>(Ac, Bc, C, DSIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    // Free CUDA streams
    CUDA_CHECK(hipStreamDestroy(stream1));
    CUDA_CHECK(hipStreamDestroy(stream2));

    // Free unified memory
    CUDA_CHECK(hipFree(A));
    CUDA_CHECK(hipFree(B));
    CUDA_CHECK(hipFree(Ac));
    CUDA_CHECK(hipFree(Bc));

    return C; // Return result (managed memory pointer)
}

int main(int argc, char const *argv[]) {
    bool check = false;
    if ( strcmp( argv[1], "-check") == 0){
        check = true;
    }
    int DSIZE;
    int print_num = 10;
    int * C;
    if (check){
        DSIZE = 10;
        C = stencilMatmul(false, 1, DSIZE);
        if (C[0] != 10)
            printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", 0,0, C[0], 10);
        if (C[1] != 42)
            printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", 0,1, C[1], 42);
        if (C[11] != 202)
            printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", 2,1, C[11], 202);
    } else{
        DSIZE = 512;
        const int radius = 3;
        C = stencilMatmul(true, radius, DSIZE);
    }

    printf("C = [\n");
    for (int i = 0; i < print_num; i++) {
        printf("     [");
        for (int j = 0; j < print_num; j++) {
            printf("%3d, ", C[DSIZE*j + i]);
        }
    printf("\b\b  ]\n");
    }
    printf("    ]\n");

    // Free unified memory for result
    CUDA_CHECK(hipFree(C));

    return 0;
}
