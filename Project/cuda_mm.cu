#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// Stencil kernel
__global__ void stencilKernel(const int* A, int* Ac, int DSIZE, int radius) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= radius && idx < DSIZE - radius && idy >= radius && idy < DSIZE - radius) {
        int temp = -A[idx * DSIZE + idy];
        for (int r = -radius; r < radius+1; r++) {
            temp += A[(idx + r) * DSIZE + idy] + A[idx * DSIZE + idy + r];
        }
        Ac[idx * DSIZE + idy] = temp;
    }
}

// Matrix multiplication kernel
__global__ void matmulKernel(const int* Ac, const int* Bc, int* C, int DSIZE) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < DSIZE && col < DSIZE) {
        int sum = 0;
        for (int k = 0; k < DSIZE; ++k) {
            sum += Ac[row * DSIZE + k] * Bc[k * DSIZE + col];
        }
        C[row * DSIZE + col] = sum;
    }
}

// Host function
int* stencilMatmul(bool isRand, int radius, const int DSIZE) {
    // Unified memory allocation
    int *A, *B, *Ac, *Bc, *C;
    CUDA_CHECK(hipMallocManaged(&A, DSIZE * DSIZE * sizeof(int)));
    CUDA_CHECK(hipMallocManaged(&B, DSIZE * DSIZE * sizeof(int)));
    CUDA_CHECK(hipMallocManaged(&Ac, DSIZE * DSIZE * sizeof(int)));
    CUDA_CHECK(hipMallocManaged(&Bc, DSIZE * DSIZE * sizeof(int)));
    CUDA_CHECK(hipMallocManaged(&C, DSIZE * DSIZE * sizeof(int)));

    // Initialize matrices
    for (int i = 0; i < DSIZE; ++i) {
        for (int j = 0; j < DSIZE; ++j) {
            A[i * DSIZE + j] = isRand ? rand() % 10 : 1;
            B[i * DSIZE + j] = isRand ? rand() % 10 : 1;
            Ac[i * DSIZE + j] = A[i * DSIZE + j];
            Bc[i * DSIZE + j] = B[i * DSIZE + j];
            C[i * DSIZE + j] = 0;
        }
    }

    // Kernel configurations
    dim3 blockDim(16, 16);
    dim3 gridDim((DSIZE + blockDim.x - 1) / blockDim.x, (DSIZE + blockDim.y - 1) / blockDim.y);

    // Launch stencil kernels
    stencilKernel<<<gridDim, blockDim>>>(A, Ac, DSIZE, radius);
    stencilKernel<<<gridDim, blockDim>>>(B, Bc, DSIZE, radius);
    CUDA_CHECK(hipDeviceSynchronize());

    // Launch matrix multiplication kernel
    matmulKernel<<<gridDim, blockDim>>>(Ac, Bc, C, DSIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    // Free unified memory
    CUDA_CHECK(hipFree(A));
    CUDA_CHECK(hipFree(B));
    CUDA_CHECK(hipFree(Ac));
    CUDA_CHECK(hipFree(Bc));

    return C; // Return result (managed memory pointer)
}

int main(int argc, char const *argv[]) {
    bool check = false;
    if ( argc > 1 && strcmp( argv[1], "-check") == 0){
        check = true;
    }
    int DSIZE;
    int print_num = 10;
    int * C;
    if (check){
        DSIZE = 10;
        C = stencilMatmul(false, 1, DSIZE);
        if (C[0] != 10)
            printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", 0,0, C[0], 10);
        if (C[1] != 42)
            printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", 0,1, C[1], 42);
        if (C[11] != 202)
            printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", 2,1, C[11], 202);
    } else{
        DSIZE = 512;
        const int radius = 3;
        C = stencilMatmul(true, radius, DSIZE);
    }

    printf("C = [\n");
    for (int i = 0; i < print_num; i++) {
        printf("     [");
        for (int j = 0; j < print_num; j++) {
            printf("%3d, ", C[DSIZE*j + i]);
        }
    printf("\b\b  ]\n");
    }
    printf("    ]\n");

    // Free unified memory for result
    CUDA_CHECK(hipFree(C));

    return 0;
}
