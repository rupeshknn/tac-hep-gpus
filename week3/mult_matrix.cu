
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

const int DSIZE = 256;
const float A_val = 3.0f;
const float B_val = 2.0f;

// error checking macro
#define cudaCheckErrors(msg)                                   \
   do {                                                        \
       hipError_t __err = hipGetLastError();                 \
       if (__err != hipSuccess) {                             \
           fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                   msg, hipGetErrorString(__err),             \
                   __FILE__, __LINE__);                        \
           fprintf(stderr, "*** FAILED - ABORTING\n");         \
           exit(1);                                            \
       }                                                       \
   } while (0)

// Square matrix multiplication on CPU : C = A * B
void matrix_mul_cpu(const float *A, const float *B, float *C, int size) {
  //FIXME:
  for (int i=0; i<size; i++){
    for (int j=0; j<size; j++){
        C[i*size+j] = 0.0;
        for (int k=0; k<size; k++){
            C[i*size+j] += A[i*size+k]*B[k*size+j];
        }
    }
  }
}

// Square matrix multiplication on GPU : C = A * B
__global__ void matrix_mul_gpu(const float *A, const float *B, float *C, int size) {

    //FIXME:
    // create thread x index
    // create thread y index
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    // Make sure we are not out of range
    if ((idx < size) && (idy < size)) {
        float temp = 0;
        for (int i = 0; i < size; i++){
            //FIXME : Add dot product of row and column
            temp += A[idx*size+i]*B[i*size+idy];
        }
        C[idx*size+idy] = temp;             
    }
}

int main() {

    float *h_A, *h_B, *h_C, *h_Ccpu, *d_A, *d_B, *d_C;
    int print_num = 3;
    // These are used for timing
    clock_t t0, t1, t2, t3;
    double t1sum=0.0;
    double t2sum=0.0;
    double t3sum=0.0;

    // start timing
    t0 = clock();

    // N*N matrices defined in 1 dimention
    // If you prefer to do this in 2-dimentions cupdate accordingly
    h_A = new float[DSIZE*DSIZE];
    h_B = new float[DSIZE*DSIZE];
    h_C = new float[DSIZE*DSIZE];
    h_Ccpu = new float[DSIZE*DSIZE];
    for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
        h_Ccpu[i] = 0;
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data from host to device
    hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
    cudaCheckErrors("Allocaiton");
    //FIXME:Add all other allocations and copies from host to device
    hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("Memory copy Host->Device");
    // Launch kernel
    // Specify the block and grid dimentions
    dim3 block(32,32);  //FIXME
    dim3 grid(DSIZE/32,DSIZE/32); //FIXME
    matrix_mul_gpu<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
    cudaCheckErrors("Kernel Launch");
    // Copy results back to host
    hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("Memory copy Device->Host");
    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. GPU Compute took %f seconds\n", t2sum);

    // FIXME
    // Excecute and time the cpu matrix multiplication function
    matrix_mul_cpu(h_A, h_B, h_Ccpu, DSIZE);

    // CPU timing
    t3 = clock();
    t3sum = ((double)(t3-t2))/CLOCKS_PER_SEC;
    printf ("Done. CPU Compute took %f seconds\n", t3sum);

    
    printf("C_GPU = [");
    for (int i = 0; i < print_num; i++) {
        printf("[ ");
        for (int j = 0; j < print_num; j++) {
            printf("%f, ", h_C[DSIZE*j + i]);
        }
        printf("]\n");
    }
    printf("]\n");

    printf("C_CPU = [");
    for (int i = 0; i < print_num; i++) {
        printf("[ ");
        for (int j = 0; j < print_num; j++) {
            printf("%f, ", h_Ccpu[DSIZE*j + i]);
        }
        printf("]\n");
    }
    printf("]\n");

    // FIXME
    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_Ccpu);
    
    return 0;

}

// for DSIZE = 256
// Done. GPU Compute took 0.326956 seconds
// Done. CPU Compute took 0.117284 seconds

// for DSIZE = 512
// Done. GPU Compute took 0.348977 seconds
// Done. CPU Compute took 1.072144 seconds